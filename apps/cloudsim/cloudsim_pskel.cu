#include "hip/hip_runtime.h"
//#define PSKEL_OMP 1
//#define PSKEL_TBB 1
#define PSKEL_CUDA 1
#define CLOUDSIM_KERNEL

#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <fstream>
#include <string>
#include <sys/stat.h>
#include <algorithm>

#include "../../pskel/include/PSkel.h"
#include "../../pskel/include/hr_time.h"

using namespace std;
using namespace PSkel;

#define WIND_X_BASE	15
#define WIND_Y_BASE	12
#define DISTURB		0.1f
#define CELL_LENGTH	0.1f
#define K           0.0243f
#define DELTAPO     0.5f
#define TAM_VETOR_FILENAME  200

struct Cloud{	
	//Args2D<double> wind_x, wind_y;
	Array2D<double> wind_x;
	Array2D<double> wind_y;
	double deltaT;
	
	Cloud(){};
	
	Cloud(int linha, int coluna){		
		//new (&wind_x) Args2D<double>(linha, coluna);
		//new (&wind_y) Args2D<double>(linha, coluna);
		new (&wind_x) Array2D<double>(linha, coluna);
		new (&wind_y) Array2D<double>(linha, coluna);
	}
};

namespace PSkel{	
__parallel__ void stencilKernel(Array2D<double> input,Array2D<double> output,Mask2D<double> mask,Cloud cloud,size_t i, size_t j){
	int numNeighbor = 0.25f;
	double sum;
	double inValue = input(i,j);
           
        double xwind = cloud.wind_x(i,j);
        double ywind = cloud.wind_y(i,j);
        int xfactor = (xwind>0)?1:-1;
        int yfactor = (ywind>0)?1:-1;
	
        sum =   (inValue - input(i-1,j) ) + (inValue - input(i,j-1) ) +
                (inValue - input(i,j+1) ) + (inValue - input(i+1,j) );
       	//sum = 4 * inValue - ( input(i-1,j) + input(i+1,j) + input(i,j+1) + input(i,j-1) );
         
        double temperaturaNeighborX = input(i,(j+xfactor));
       	double temperaturaNeighborY = input((i+yfactor),j);
        
        double componenteVentoY = yfactor * ywind;
     	double componenteVentoX = xfactor * xwind;
        
        double temp_wind = (-componenteVentoX * ((inValue - temperaturaNeighborX)*10.0f)) -
                          ( componenteVentoY * ((inValue - temperaturaNeighborY)*10.0f));
        	
		/*
		double temp_wind = 0.0f;
		int height=input.getHeight();
        int width=input.getWidth();
        if ( (j == 0) && (i == 0) ) {
            sum = (inValue - input(i+1,j) ) +
                  (inValue - input(i,j+1) );
            numNeighbor = 2;
        }	//	Corner 2	
        else if ((j == 0) && (i == width-1)) {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i,j+1) );
            numNeighbor = 2;
        }	//	Corner 3	
        else if ((j == height-1) && (i == width-1)) {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i,j-1) );
            numNeighbor = 2;
        }	//	Corner 4	
        else if ((j == height-1) && (i == 0)) {
            sum = (inValue - input(i,j-1) ) +
                  (inValue - input(i+1,j) );
            numNeighbor = 2;
        }	//	Edge 1	
        else if (j == 0) {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i+1,j) ) +
                  (inValue - input(i,j+1) );
            numNeighbor = 3;
        }	//	Edge 2	
        else if (i == width-1) {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i,j-1) ) +
                  (inValue - input(i,j+1) );
            numNeighbor = 3;
        }	//	Edge 3	
        else if (j == height-1) {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i,j-1) ) +
                  (inValue - input(i+1,j) );
            numNeighbor = 3;
        }	//	Edge 4	
        else if (i == 0) {
            sum = (inValue - input(i,j-1) ) +
                  (inValue - input(i,j+1) ) +
                  (inValue - input(i+1,j) );
            numNeighbor = 3;
        }	//	Inside the cloud  
        else {
            sum = (inValue - input(i-1,j) ) +
                  (inValue - input(i,j-1) ) +
                  (inValue - input(i,j+1) ) +
                  (inValue - input(i+1,j) );
            numNeighbor = 4;
            
            double xwind = cloud.wind_x(i,j);
            double ywind = cloud.wind_y(i,j);
            int xfactor = (xwind>0)?1:-1;
            int yfactor = (ywind>0)?1:-1;

            double temperaturaNeighborX = input(i,(j+xfactor));
            double componenteVentoX = xfactor * xwind;
            double temperaturaNeighborY = input((i+yfactor),j);
            double componenteVentoY = yfactor * ywind;
        
            temp_wind = (-componenteVentoX * ((inValue - temperaturaNeighborX)/CELL_LENGTH)) -
                        ( componenteVentoY * ((inValue - temperaturaNeighborY)/CELL_LENGTH));
            
        }*/
        double temperatura_conducao = -K*(sum * numNeighbor) * cloud.deltaT;
        double result = inValue + temperatura_conducao;
        output(i,j) = result + temp_wind * cloud.deltaT;

		/*
        	for( int m = 0; m < mask.size ; m++ ){
			double temperatura_vizinho = mask.get(m,input,i,j);
			int factor = (temperatura_vizinho==0)?0:1;
			sum += factor*(inValue - temperatura_vizinho);
			numNeighbor += factor;
		}
		
        		
		double temperatura_conducao = -K*(sum / numNeighbor)*cloud.deltaT;
		
		double result = inValue + temperatura_conducao;
		
		double xwind = cloud.wind_x(i,j);
		double ywind = cloud.wind_y(i,j);
		int xfactor = (xwind>0)?3:1;
		int yfactor = (ywind>0)?2:0;

		double temperaturaNeighborX = mask.get(xfactor,input,i,j);
		double componenteVentoX = (xfactor-2)*xwind;
		double temperaturaNeighborY = mask.get(yfactor,input,i,j);
		double componenteVentoY = (yfactor-1)*ywind;
		
		double temp_wind = (-componenteVentoX * ((inValue - temperaturaNeighborX)/CELL_LENGTH)) -(componenteVentoY * ((inValue - temperaturaNeighborY)/CELL_LENGTH));
		
		output(i,j) = result + ((numNeighbor==4)?(temp_wind*cloud.deltaT):0.0f);
        */
	}	
}

/* Convert Celsius to Kelvin */
double Convert_Celsius_To_Kelvin(double number_celsius)
{
	double number_kelvin;
	number_kelvin = number_celsius + 273.15f;
	return number_kelvin;
}

/* Convert Pressure(hPa) to Pressure(mmHg) */
double Convert_hPa_To_mmHg(double number_hpa)
{
	double number_mmHg;
	number_mmHg = number_hpa * 0.750062f;

	return number_mmHg;
}

/* Convert Pressure Millibars to mmHg */
double Convert_milibars_To_mmHg(double number_milibars)
{
	double number_mmHg;
	number_mmHg = number_milibars * 0.750062f;

	return number_mmHg;
}

/* Calculate RPV */
double CalculateRPV(double temperature_Kelvin, double pressure_mmHg)
{
	double realPressureVapor; //e
	double PsychrometricConstant = 6.7f * powf(10,-4); //A
	double PsychrometricDepression = 1.2f; //(t - tu) in ºC
	double esu = pow(10, ((-2937.4f / temperature_Kelvin) - 4.9283f * log10(temperature_Kelvin) + 23.5470f)); //10 ^ (-2937,4 / t - 4,9283 log t + 23,5470)
	realPressureVapor = Convert_milibars_To_mmHg(esu) - (PsychrometricConstant * pressure_mmHg * PsychrometricDepression);

	return realPressureVapor;
}

/* Calculate Dew Point */
double CalculateDewPoint(double temperature_Kelvin, double pressure_mmHg)
{
	double dewPoint; //TD
	double realPressureVapor = CalculateRPV(temperature_Kelvin, pressure_mmHg); //e
	dewPoint = (186.4905f - 237.3f * log10(realPressureVapor)) / (log10(realPressureVapor) -8.2859f);

	return dewPoint;
}

int main(int argc, char **argv){
	int linha, coluna, i, j, timeTileSize,numero_iteracoes, raio_nuvem, menu_option, GPUBlockSizeX, GPUBlockSizeY, numCPUThreads;
	double temperaturaAtmosferica, pressaoAtmosferica, pontoOrvalho, limInfPO, limSupPO, deltaT, GPUTime;
	//double alturaNuvem;
    //int write_step;
	if (argc != 10){
		printf ("Wrong number of parameters.\n");
		//printf ("Usage: cloudsim Numero_Iteraoes Linha Coluna Raio_Nuvem Temperatura_Atmosferica Altura_Nuvem Pressao_Atmosferica Delta_T GPUTIME GPUBLOCKS CPUTHREADS Menu_Option Write_Step\n");
		printf ("Usage: cloudsim WIDTH HEIGHT ITERATIONS TIME_TILE_SIZE GPUTIME GPUBLOCK_X GPU_BLOCK_Y CPUTHREADS OUTPUT_WRITE_FLAG\n");
		exit (-1);
	}
	//20 -3 5.0 700.0 0.001 1.0 32 12 0 10
	
	coluna = atoi(argv[1]);
	linha = atoi(argv[2]);
	numero_iteracoes = atoi(argv[3]);
	timeTileSize = atoi(argv[4]);
	GPUTime = atof(argv[5]);
	GPUBlockSizeX = atoi(argv[6]);
	GPUBlockSizeY = atoi(argv[7]);
	numCPUThreads = atoi(argv[8]);
	menu_option = atoi(argv[9]);
	
	raio_nuvem = 20; 				//atoi(argv[4]);
	temperaturaAtmosferica = -3.0f; 	//atof(argv[5]);
	//alturaNuvem = 5.0; 				//atof(argv[6]);
	pressaoAtmosferica =  700.0f;		//atof(argv[7]);
	deltaT = 0.01f;					//atof(argv[8]);
	
	//numThreads = numCPUThreads;
	//write_step = 10;				//atoi(argv[13]);
	
	//global_write_step = write_step;
	pontoOrvalho = CalculateDewPoint(Convert_Celsius_To_Kelvin(temperaturaAtmosferica), Convert_hPa_To_mmHg(pressaoAtmosferica));
	limInfPO = pontoOrvalho - DELTAPO;
	limSupPO = pontoOrvalho + DELTAPO;
	//char maindir[30];
	//char dirname[TAM_VETOR_FILENAME];
	//char dirMatrix_temp[TAM_VETOR_FILENAME];
	//char dirMatrix_stat[TAM_VETOR_FILENAME];
	//char dirMatrix_windX[TAM_VETOR_FILENAME];
	//char dirMatrix_windY[TAM_VETOR_FILENAME];
	//double start_time = 0;
	//double end_time = 0;
		
	Array2D<double> inputGrid(coluna, linha);
	Array2D<double> outputGrid(coluna, linha);
	Mask2D<double> mask(4);
	
	mask.set(0,0,1);
	mask.set(1,1,0);
	mask.set(2,0,-1);
	mask.set(3,-1,0);
	
	Cloud cloud(linha,coluna);
	cloud.deltaT = deltaT;
	
	//omp_set_num_threads(numCPUThreads);

	/* Inicialização da matriz de entrada com a temperatura ambiente */
	//#pragma omp parallel for private (i,j)
	for (i = 0; i < linha; i++){		
		for (j = 0; j < coluna; j++){
			inputGrid(i,j) = temperaturaAtmosferica;
			//outputGrid(i,j) = temperaturaAtmosferica;
		}
	}	
	/* Inicialização dos ventos Latitudinal(Wind_X) e Longitudinal(Wind_Y) */
    	srand(1234);
	for( i = 0; i < linha; i++ ){
		for(j = 0; j < coluna; j++ ){			
			cloud.wind_x(i,j) = (WIND_X_BASE - DISTURB) + (double)rand()/RAND_MAX * 2 * DISTURB;
			cloud.wind_y(i,j) = (WIND_Y_BASE - DISTURB) + (double)rand()/RAND_MAX * 2 * DISTURB;		
		}
	}
	
	//Forcing copy
	if(GPUTime > 0){
		cloud.wind_x.deviceAlloc();
		cloud.wind_x.copyToDevice();
		cloud.wind_y.deviceAlloc();
		cloud.wind_y.copyToDevice();	
	}
					
	/* Inicialização de uma nuvem no centro da matriz de entrada */
	srand(1);
	int y, x0 = linha/2, y0 = coluna/2;
	for(i = x0 - raio_nuvem; i < x0 + raio_nuvem; i++){
		 // Equação da circunferencia: (x0 - x)² + (y0 - y)² = r²
		y = (int)((floor(sqrt(pow((double)raio_nuvem, 2.0) - pow(((double)x0 - (double)i), 2)) - y0) * -1));
		for(int j = y0 + (y0 - y); j >= y; j--){
			inputGrid(i,j) = limInfPO + (double)rand()/RAND_MAX * (limSupPO - limInfPO);
			//outputGrid(i,j) = limInfPO + (double)rand()/RAND_MAX * (limSupPO - limInfPO);
		}
	}
	
    #ifdef PSKEL_PAPI
		if(GPUTime < 1.0)
			PSkelPAPI::init(PSkelPAPI::CPU);
	#endif
    
    	hr_timer_t timer;
	hrt_start(&timer);
    
	Stencil2D<Array2D<double>, Mask2D<double>, Cloud> stencilCloud(inputGrid, outputGrid, mask, cloud);
	
	if(GPUTime == 0.0){
		//cout<<"Running Iterative CPU"<<endl;
		//if(numCPUThreads == 1)
		//	stencilCloud.runSequential();
		//else
		//	stencilCloud.runIterativeCPU(numero_iteracoes, numCPUThreads);
            
        #ifdef PSKEL_PAPI
            for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
			PSkelPAPI::papi_start(PSkelPAPI::CPU,i);
		#endif
			//stencilCloud.runIterativeCPU(numero_iteracoes, numCPUThreads);	
		#ifdef PSKEL_PAPI
			PSkelPAPI::papi_stop(PSkelPAPI::CPU,i);
            }
		#endif
	}
	else if(GPUTime == 1.0){
		//Forcing copy
		//cloud.wind_x.deviceAlloc();
		//cloud.wind_x.copyToDevice();
		//cloud.wind_y.deviceAlloc();
		//cloud.wind_y.copyToDevice();
        #ifdef PSKEL_SHARED	
            stencilCloud.runIterativeGPU(numero_iteracoes, timeTileSize,GPUBlockSizeX, GPUBlockSizeY);
        #else
            stencilCloud.runIterativeGPU(numero_iteracoes, GPUBlockSizeX, GPUBlockSizeY);
        #endif
	}
	else{
		//stencilCloud.runIterativePartition(numero_iteracoes, GPUTime, numCPUThreads,GPUBlockSizeX, GPUBlockSizeY);
	}
	
	hrt_stop(&timer);
    
	if(menu_option == 1){
		cout.precision(6);
		cout<<std::fixed;
		cout<<"INPUT"<<endl;
		for( i = 0; i < linha; i++ ){
                	for(j = 0; j < coluna; j++ ){
				cout<<inputGrid(i,j)<<"\t";
			}
			cout<<endl;
		}
	
	/*
		for(int i=10; i<coluna;i+=10){
			cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t\t("<<coluna-i<<","<<linha-i<<") = "<<inputGrid(coluna-i,linha-i)<<endl;
		}
		cout<<endl;
		
		cout<<"OUTPUT"<<endl;
		for(int i=10; i<coluna;i+=10){
			cout<<"("<<i<<","<<i<<") = "<<outputGrid(i,i)<<"\t\t("<<coluna-i<<","<<linha-i<<") = "<<outputGrid(coluna-i,linha-i)<<endl;
		}
		cout<<endl;
		*/
		cout<<"OUTPUT"<<endl;
		for(int h = 0; h < linha; ++h){		
			for(int w = 0; w < coluna; ++w){
				cout<<outputGrid(h,w)<<"\t";
			}
			cout<<endl;
		}
	}
	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0){
			PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
			PSkelPAPI::shutdown();
		}
	#endif
	cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;
	
	return 0;
}

