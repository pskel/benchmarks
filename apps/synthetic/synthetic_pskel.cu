#include "hip/hip_runtime.h"
#define TIME 1
//#define PSKEL_LOGMODE 1
//#define TBB_USE_DEBUG 1

#define PSKEL_CUDA

#include "PSkel.h"
#include "hr_time.h"
#include <omp.h>
#include <fstream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <sstream> 
#include <cmath>
#include <cassert>


#ifndef PSKEL_NEUMAN
	#ifndef PSKEL_MOORE
		#define PSKEL_NEUMAN
	#endif
#else
#ifndef PSKEL_MOORE
	#ifndef PSKEL_NEUMAN
		#define PSKEL_MOORE
	#endif
#endif
#endif


using namespace std;
using namespace PSkel;

struct Arguments{
	//int neighborhood, radius, numAdd, numSub, numMult, numDiv, numPow, numSqrt, numFma;
	int numAdd, numMult,radius;

	Arguments(){
		//neighborhood = 0;
		radius = 2;
		numAdd = 10;
		//numSub = 0;
		numMult = 0;
		//numDiv = 0;
		//numPow = 0;
		//numSqrt = 0;
		//numFma = 0;
	}

	//Arguments(int nb, int r, int nAdd, int nSub, int nMult, int nDiv, int nPow, int nSqrt, int nFma){
    	Arguments(int r, int nAdd, int nMult){
		//neighborhood = nb;
		radius = r;
		numAdd = nAdd;
		//numSub = nSub;
		numMult = nMult;
		//numDiv = nDiv;
		//numPow = nPow;
		//numSqrt = nSqrt;
		//numFma = nFma;
	}
};

namespace PSkel{
	#ifdef PSKEL_INT
	__parallel__ void stencilKernel(Array2D<int> input,Array2D<int> output,Mask2D<int> mask,Arguments args, size_t h, size_t w){
		int returnValue = input(h,w);
	#else
	__parallel__ void stencilKernel(Array2D<float> input,Array2D<float> output,Mask2D<float> mask,Arguments args, size_t h, size_t w){
		float  returnValue = input(h,w);
	#endif

		int loopControl;
		int opControl;
		int i,j,k,fim,ini;

		loopControl = args.numAdd > 0 ? (args.numAdd-1)/mask.size + 1 : 0; //estava assim originalmente
		//opControl = args.numAdd > mask.size ? mask.size : args.numAdd;
		opControl = args.numAdd > 0 ? args.radius : 0;


		
		#ifdef PSKEL_NEUMAN
		for(int i = 0; i < loopControl; i++){    
        		fim = 0;
        		ini = 0;
        		k = 0;
        		for (j = -opControl; j <= 0; j++) {
            			for(k = ini; k <= fim; k++){
                			//if(j != 0 || k !=0){
                                       		returnValue = returnValue + input(h+j,w+k);	
                			//}
            			}
            			ini--;
            			fim++;                
        		}        
        		ini+=2;
        		fim-=2;
        
        		for(j = 1; j <= opControl; j++){
             			for(k = ini; k <= fim; k++){
                	      		returnValue = returnValue + input(h+j,w+k);		
				}
            			ini++;
            			fim--;
        		}
    		}

		#else
		#ifdef PSKEL_MOORE
		
		//Adição
		//loopControl = ceil(float(args.numAdd)/float(mask.size));
		//loopControl = args.numAdd > 0 ? (args.numAdd-1)/mask.size + 1 : 0; //estava assim originalmente
		//opControl = args.numAdd > mask.size ? mask.size : args.numAdd;
		//opControl = args.numAdd > 0 ? args.radius : 0;

		//loopControl = loopControl/2;
		//printf("Executing ADD loopControl: %d opControl: %d\n",loopControl,opControl);
		for(i = 0; i<loopControl; i++){
		//for(int i = -loopControl; i <= loopControl; i++){
			for(j = -opControl;j <= opControl; j++){
				for(k = -opControl; k <= opControl; k++){  //for(int k = 0; k < opControl; k++){
					//returnValue = returnValue + mask.get(j,input,h,w);
					//returnValue = returnValue + mask.getWeight(j);
					returnValue = returnValue + input(h+j,w+k);
				}
			}
		}
		#endif
		#endif
		

		//Multiplicação
		//loopControl = ceil(float(args.numMult)/float(mask.size));
		loopControl = args.numMult > 0 ? (args.numMult-1)/mask.size + 1 : 0;
		//opControl = args.numMult > mask.size ? mask.size: args.numMult;
		opControl = args.numMult > 0 ? args.radius : 0;
		
		//loopControl = loopControl/2;
		//printf("Executing MULT loopControl: %d opControl: %d\n",loopControl,opControl);

		#ifdef PSKEL_NEUMAN
		for(i = 0; i < loopControl; i++){    
        		fim = 0;
        		ini = 0;
        		k = 0;
        		for (j = -opControl; j <= 0; j++) {
            			for(k = ini; k<= fim; k++){
                			//if(j != 0 || k !=0){
                                       		returnValue = returnValue * input(h+j,w+k);	
                			//}
            			}
            			ini--;
            			fim++;                
        		}        
        		ini+=2;
        		fim-=2;
        
        		for(j = 1; j <= opControl; j++){
             			for(k = ini; k <= fim; k++){
                	      		returnValue = returnValue * input(h+j,w+k);		
				}
            			ini++;
            			fim--;
        		}
    		}

		#else
		#ifdef PSKEL_MOORE
		for(i = 0; i<loopControl; i++){
			for(j = -opControl; j <= opControl; j++){
				for(k = -opControl; k <= opControl; k++){
					//returnValue = returnValue * mask.get(j,input,h,w);
					//returnValue = returnValue * mask.getWeight(j);
					returnValue = returnValue * input(h+j,w+k);
				}
			}
		}		
		#endif
		#endif
		
		//Divisao
		/*loopControl = (args.numDiv-1)/mask.size + 1;
		opControl = args.numDiv>mask.size?mask.size:args.numDiv;
		for(int i = 0; i<loopControl; i++){
			for(int j = 0; j<opControl; j++){
				returnValue = returnValue / mask.get(j,input,h,w);
			}
		}

		*/

        	output(h,w) = returnValue;
		
	}
}

int main(int argc, char **argv){
    //hr_timer_t timer_a;
    //hrt_start(&timer_a);
    
    int width, height, iterations, maskType,maskRange,GPUBlockSize, numCPUThreads, maskSize,writeToFile;

    int nAdd, nMult;
    //int nSub, nDiv, nPow, nFma;
    float GPUTime;

    if (argc != 11){
        printf ("Wrong number of parameters.\n");
        //printf ("Usage: synthetic WIDTH HEIGHT ITERATIONS GPUTIME GPUBLOCKS CPUTHREADS MASKTYPE MASKRANGE NumADDS NumSUBS NumMults NumDivs NumPows NumSqrts NumFmas OUTPUT_WRITE_FLAG\n");

        printf ("Usage: synthetic WIDTH HEIGHT ITERATIONS GPUTIME GPUBLOCKS CPUTHREADS MASKRANGE NumADDS NumMults OUTPUT_WRITE_FLAG\n"); //Masktype is now defined from #ifdef
        exit(-1);
    }

    width = atoi (argv[1]);
    height = atoi (argv[2]);
    iterations = atoi (argv[3]);
    GPUTime = atof(argv[4]);
    GPUBlockSize = atoi(argv[5]);
    numCPUThreads = atoi(argv[6]);
    
    #ifdef PSKEL_NEUMAN
    maskType = 0;
    //cout<<"Neuman"<<endl;
    #else
    maskType = 1;
    //cout<<"Moore"<<endl;
    #endif

    maskRange = atoi (argv[7]); 
    nAdd = atoi(argv[8]) ;
    //nSub = 0; //atoi(argv[10]);
    nMult = atoi(argv[9]) ;
    //nDiv = 0; //atoi(argv[12]);
    //nPow = 0; //atoi(argv[13]);
    //nSqrt = 0; //atoi(argv[14]);
    //nFma = 0; //atoi(argv[15]);
    writeToFile = atoi(argv[10]);
    
    if(nAdd == 0 && nMult == 0){
	printf("The number of Adds and Mults are 0!\n");
	exit(-1);
    }

    #ifdef PSKEL_INT
    Array2D<int> inputGrid(width, height);
    Array2D<int> outputGrid(width, height);
    #else
    Array2D<float> inputGrid(width, height);
    Array2D<float> outputGrid(width, height);	
    #endif

    /*for(int h=0; h<inputGrid.getHeight(); h++)
        for(int w=0; w<inputGrid.getWidth(); w++)
            inputGrid(h,w) = h*inputGrid.getWidth()+w;
    */

    #pragma omp parallel num_threads(numCPUThreads)
    {
        unsigned int seed = 1234 + 17 *  omp_get_thread_num();
        #pragma omp for
        for (int x = 0; x < height; x++){
            for (int y = 0; y < width; y++){		
                //#ifdef PSKEL_INT
                //inputGrid(x,y) = 1 + rand()%99;
                //outputGrid(x,y) = 1;
            //	#else
                inputGrid(x,y) = (1.0 + rand_r(&seed)%9) + 1.0/(1+rand_r(&seed)%100);
                outputGrid(x,y) = 1.0;
            //	#endif
            }
        }
    }

    //Calculate the mask size based on neighborhood type. 0 (zero) for Von Neumann >1 (more than one) for Moore.
    if(maskType == 0){
        //Neumann number 2r(r+1)+1,
        //maskSize = ((2 * args.radius)*args.radius) + (2 * args.radius) + 1;
        maskSize = 1 + ((2 * maskRange ) * ( maskRange + 1));		
    }else{ 
       //Moore (2r + 1)^2
        maskSize = (2 * maskRange + 1) * (2 * maskRange + 1);
    }

    //maskSize = number of cells in neighborhood - 1 (the center cell);
    #ifdef PSKEL_INT
    Mask2D<int> mask(maskSize - 1,1);
    #else
    Mask2D<float> mask(maskSize - 1,1.0);
    #endif

    nAdd = nAdd  * (maskSize - 1);
    //nSub = 0; //atoi(argv[10]);
    nMult = nMult * (maskSize - 1);
   //Arguments args(maskType, maskRange, nAdd, nSub, nMult, nDiv, nPow, nSqrt, nFma);

    Arguments args(maskRange,nAdd, nMult);
    srand(1234);

    //cout<<"MASK VALUES"<<endl;
    if(maskType == 0){
        //Set mask for Neumann neighborhood
        int idx = 0;        
        int fim = 0;
        int ini = 0;
        int w = 0;
        int h;

        for (h = -maskRange; h <= 0; h++) {
            for(w = ini; w <= fim; w++){
                if(h != 0 || w !=0){
                    float weight = 0.1*(1+rand()%8);
                    #ifdef PSKEL_INT
                        weight += 1 + rand()%3;
                    #endif		
                    mask.set(idx, h, w, weight);

                    //cout<<"["<<idx<<"] = "<<h<<","<<w<<" "<<weight<<endl;
                //cout << idx <<", "<< h <<", "<< w <<"\n";
                idx++;
                }
            }

            ini --;
            fim ++;                
        }        
        ini+=2;
        fim-=2;
        
        for(h = 1; h <= maskRange; h++){
             for(w = ini; w <= fim; w++){
                float weight = 0.1*(1+rand()%8);
                    #ifdef PSKEL_INT
                        weight += 1 + rand()%3;
                    #endif		

                mask.set(idx, h, w, weight);
                //cout<<"["<<idx<<"] = "<<h<<","<<w<< " "<<weight<<endl;
                idx++;
            }
            ini++;
            fim--;
        }
    }
    else{ 
        //Set mask for Moore neighborhood
        int idx = 0;
        int h, w;

        for(h = -maskRange; h <= maskRange; h++){
            for(w = -maskRange; w <= maskRange; w ++){
                if(h != 0 || w != 0){
                    float weight = 0.1*(1+rand()%8);
                    #ifdef PSKEL_INT
                        weight += 1 + rand()%3;
                    #endif		
                    mask.set(idx, h, w, weight);
                    //cout<<"["<<idx<<"] = "<<h<<","<<w<< " "<<weight<<endl;
                    idx ++;
                }
            }
        }
    }
    	
    cout <<"\n";
    cout << "Width: " << width << "; Height: " << height << ";\n";
    cout << "Iterations: " << iterations << endl;	
    cout << "MaskType: " << maskType << endl;
    cout << "MaskRange: " << maskRange << endl;
    cout << "Neighbors: "	 << mask.size << endl;
    cout << "GPU Time: " << GPUTime << endl;
    cout << "GPU Block size: " << GPUBlockSize << endl;
    cout << "CPU Threads: " << numCPUThreads << endl;
    cout << "Num Add: " << args.numAdd << endl;
    //cout << "Num Sub: " << args.numSub << endl;
    cout << "Num Mult: " << args.numMult << endl;
    //cout << "Num Div: " << args.numDiv << endl;	
    //cout << "Num Pow: " << args.numPow << endl;
    //cout << "Num Sqrt: " << args.numSqrt << endl;
    //cout << "Num Fma: " << args.numFma << endl;
    cout <<"\n";
    
            
    #ifdef PSKEL_INT
    Stencil2D<Array2D<int>, Mask2D<int>, Arguments> synthetic(inputGrid, outputGrid, mask, args);
    #else
    Stencil2D<Array2D<float>, Mask2D<float>, Arguments> synthetic(inputGrid, outputGrid, mask, args);
    #endif

    #ifdef PSKEL_PAPI
        if(GPUTime < 1.0)
            PSkelPAPI::init(PSkelPAPI::CPU);
    #endif

    //hrt_stop(&timer_a);
    //cout << "Init time: " << hrt_elapsed_time(&timer_a) << endl;
    //cout << "Executing" << endl;
    
    hr_timer_t timer;
    //double t1,t2;
    //t1 = omp_get_wtime();
    hrt_start(&timer);

    if(GPUTime == 0.0){
        #ifdef PSKEL_PAPI
        for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
            //cout << "Running iteration " << i << endl;
	    PSkelPAPI::papi_start(PSkelPAPI::CPU,i);
            synthetic.runIterativeCPU(iterations, numCPUThreads);	
	    PSkelPAPI::papi_stop(PSkelPAPI::CPU,i);
        }
        #else
            //cout<<"Running Iterative CPU"<<endl;
            synthetic.runIterativeCPU(iterations, numCPUThreads);	
        #endif
    }

    else if(GPUTime == 1.0){
        synthetic.runIterativeGPU(iterations, GPUBlockSize,GPUBlockSize);
    }
    else{
        #ifdef PSKEL_PAPI
        for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
            synthetic.runIterativePartition(iterations, GPUTime, numCPUThreads,GPUBlockSize,i);
        }
        #else
            synthetic.runIterativePartition(iterations, GPUTime, numCPUThreads,GPUBlockSize,GPUBlockSize);
        #endif
    }
    //t2 = omp_get_wtime();
    hrt_stop(&timer);		

    #ifdef PSKEL_PAPI
        hipDeviceReset();
        if(GPUTime < 1.0){
            PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
            PSkelPAPI::shutdown();
        }
    #endif

    cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;
    //cout << "Exec_time_omp\t" << t2-t1 << endl;

    if(writeToFile == 1){
        cout.precision(12);
        cout<<"INPUT"<<endl;
        for(int i=10; i<width;i+=10){
            cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t\t("<<width-i<<","<<height-i<<") = "<<inputGrid(width-i,height-i)<<endl;
        }
        cout<<endl;
        
        cout<<"OUTPUT"<<endl;
        
        for(int i=10; i<width;i+=10){
            cout<<"("<<i<<","<<i<<") = "<<outputGrid(i,i)<<"\t\t("<<width-i<<","<<height-i<<") = "<<outputGrid(width-i,height-i)<<endl;
        }
        cout<<endl;
    }

    return 0;
}
