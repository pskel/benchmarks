//#define PSKEL_LOGMODE 1

#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <fstream>

//#define PSKEL_SHARED_MASK
#define PSKEL_CUDA
//#define GOL_KERNEL
//#define PSKEL_PAPI
//#define PSKEL_PAPI_DEBUG
#ifndef PSKEL_OMP
        #ifndef PSKEL_TBB
                #define PSKEL_OMP
                #undef PSKEL_TBB
        #endif
#else
#ifndef PSKEL_TBB
        #ifndef PSKEL_OMP
                #define PSKEL_TBB
                #undef PSKEL_OMP
        #endif
#endif
#endif


#include "PSkel.h"
//#include "hr_time.h"
//#include "wb.h"

using namespace std;
using namespace PSkel;

namespace PSkel{
__parallel__ void stencilKernel(Array2D<bool> &input, Array2D<bool> &output, Mask2D<bool> &mask, bool args, size_t i, size_t j){
    int neighbors =  input(i-1,j-1) + input(i-1,j) + input(i-1,j+1)  +
                     input(i,j-1)   + input(i,j+1) + 
		     input(i+1,j-1) + input(i+1,j) + input(i+1,j+1);
                      
    bool central = input(i,j);
    //printf("%d,%d\n",i,j);
    //int neighbors = mask.get(0,input,i,j) + mask.get(1,input,i,j) + mask.get(2,input,i,j) +

    //	              mask.get(3,input,i,j) + mask.get(4,input,i,j) + mask.get(5,input,i,j) +
    //		      mask.get(6,input,i,j) + mask.get(7,input,i,j);
    
    //int neighbors =  input(i-1,j-1) + input(i-1,j) + input(i-1,j+1)  +
    //                 input(i+1,j-1) + input(i+1,j) + input(i+1,j+1)  + 
    //                 input(i,j-1)   + input(i,j+1) ;
    /*
    bool neighbors = 0;
    bool height=input.getHeight();
    bool width=input.getWidth();
    
    if ( (j == 0) && (i == 0) ) { //	Corner 1	
        neighbors = input(i+1,j) + input(i,j+1) + input (i+1,j+1);
    }	//	Corner 2	
    else if ((j == 0) && (i == width-1)) {
        neighbors = input(i-1,j) + input(i,j+1) + input(i-1,j+1);
    }	//	Corner 3	
    else if ((j == height-1) && (i == width-1)) {
        neighbors = input(i-1,j) + input(i,j-1) + input(i-1,j-1);
    }	//	Corner 4	
    else if ((j == height-1) && (i == 0)) {
        neighbors = input(i,j-1) + input(i+1,j) + input(i+1,j-1);
    }	//	Edge 1	
    else if (j == 0) {
        neighbors = input(i-1,j) + input(i+1,j) + input(i-1,j+1) + input(i,j+1) + input(i+1,j+1);
    }	//	Edge 2	
    else if (i == width-1) {
        neighbors = input(i,j-1) +  input(i-1,j-1) + input(i-1,j) +  input(i-1,j+1) + input(i,j+1);
    }	//Edge 3	
    else if (j == height-1) {
        neighbors = input(i-1,j-1) + input(i,j-1) + input(i+1,j-1) + input(i-1,j) + input(i+1,j);
    }	//Edge 4
    else if (i == 0) {
        neighbors = input(i,j-1) + input(i+1,j-1) + input(i+1,j) + input(i,j+1) + input(i+1,j+1);
    }	//Inside the grid
    else {
        neighbors =  input(i-1,j-1) + input(i-1,j) + input(i-1,j+1)  +
                     input(i+1,j-1) + input(i+1,j) + input(i+1,j+1)  + 
                     input(i,j-1)   + input(i,j+1) ;
    }
    */ 
    output(i,j) = (neighbors == 3 || (neighbors == 2 && central))?1:0;
        
    }
}

int main(int argc, char **argv){
	int width, height, T_MAX,timeTileSize,GPUBlockSizeX, GPUBlockSizeY, numCPUThreads,verbose;
	float GPUTime;

	if (argc != 10){
		printf ("Wrong number of parameters.\n");
		printf ("Usage: gol WIDTH HEIGHT ITERATIONS TIME_TILE_SIZE GPUPERCENT GPUBLOCKS_X GPUBLOCKS_Y CPUTHREADS VERBOSE\n");
		exit (-1);
	}

	width = atoi (argv[1]);
	height = atoi (argv[2]);
	T_MAX = atoi(argv[3]);
    	timeTileSize = atoi(argv[4]);
	GPUTime = atof(argv[5]);
	GPUBlockSizeX = atoi(argv[6]);
	GPUBlockSizeY = atoi(argv[7]);
	numCPUThreads = atoi(argv[8]);
	verbose = atoi(argv[9]);
	
	Array2D<bool> inputGrid(width, height);
	Array2D<bool> outputGrid(width, height);
	Mask2D<bool> mask(8);
	
	mask.set(0,-1,-1);	mask.set(1,-1,0);	mask.set(2,-1,1);
	mask.set(3,0,-1);						mask.set(4,0,1);
	mask.set(5,1,-1);	mask.set(6,1,0);	mask.set(7,1,1);
		
	//omp_set_num_threads(numCPUThreads);

	//srand(123456789);
	#pragma omp parallel num_threads(numCPUThreads)
	{
	unsigned int seed = 25234 + 17 * omp_get_thread_num();
	#pragma omp for
    	for(int h = 0; h < height; h++){		
       		for(int w = 0; w < width; w++){
      			inputGrid(h,w) = (bool) (rand_r(&seed)%2) ;            
            		//outputGrid(i,j) =  inputGrid(i,j);
		}
	}
	}

	if(verbose){
 		cout<<"INPUT"<<endl;
                for(size_t h = 0; h < height; h++){             
                        for(size_t w = 0; w < width; w++){
                                cout<<inputGrid(h,w);
                        }
                        cout<<endl;
                }
	}

	#ifdef PSKEL_PAPI
	if(GPUTime < 1.0)
		PSkelPAPI::init(PSkelPAPI::CPU);
	else 
		PSkelPAPI::init(PSkelPAPI::NVML);
	#endif	
	
	//hr_timer_t timer;
	//hrt_start(&timer);
	//wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
	Stencil2D<Array2D<bool>, Mask2D<bool>, bool> stencil(inputGrid, outputGrid, mask, 0);
	
	if(GPUTime == 0.0){
		//jacobi.runIterativeCPU(T_MAX, numCPUThreads);
		//#ifdef PSKEL_PAPI
		//	for(unsigned bool i=0;i<NUM_GROUPS_CPU;i++){
		//		//cout << "Running iteration " << i << endl;
		//		stencil.runIterativeCPU(T_MAX, numCPUThreads, i);	
		//	}
		//#else
			//cout<<"Running Iterative CPU"<<endl;
		
		#ifdef PSKEL_PAPI
		//for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
		PSkelPAPI::papi_start(PSkelPAPI::CPU,5);
		#endif
		stencil.runIterativeCPU(T_MAX, numCPUThreads);	
		#ifdef PSKEL_PAPI
		PSkelPAPI::papi_stop(PSkelPAPI::CPU,5);
		//}
		#endif
	}
	else if(GPUTime == 1.0){
		#ifdef PSKEL_PAPI
        PSkelPAPI::papi_start(PSkelPAPI::NVML,0);
        #endif
        #ifdef PSKEL_SHARED
		//stencil.runIterativeGPU(T_MAX,timeTileSizei,GPUBlockSizeX, GPUBlockSizeY);
        #else
        	stencil.runIterativeGPU(T_MAX,GPUBlockSizeX, GPUBlockSizeY);
        #endif
		#ifdef PSKEL_PAPI
        PSkelPAPI::papi_stop(PSkelPAPI::NVML,0);
        #endif

	}
	else{
		stencil.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX, GPUBlockSizeY);
		/*
        	#ifdef PSKEL_PAPI
			for(unsigned bool i=0;i<NUM_GROUPS_CPU;i++){
				stencil.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX,i);
			}
		#else
			//stencil.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX);
		#endif
        */
	}
	
	
	//wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
	//hrt_stop(&timer);

	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0){
			PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
		}
		else{
			PSkelPAPI::print_profile_values(PSkelPAPI::NVML);
		}
		PSkelPAPI::shutdown();
	#endif
    
    if(verbose){		
		//cout<<setprecision(6);
		//cout<<fixed;
		//cout<<"INPUT"<<endl;
		/*for(bool i=0; i<width;i+=10){
            
			cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t\t(";
            cout<<width-i<<","<<height-i<<") = "<<inputGrid(height-i,width-i)<<endl;
		}
		cout<<endl;
        */
        	//for(size_t h = 0; h < height; h++){		
		//	for(size_t w = 0; w < width; w++){
		//		cout<<inputGrid(h,w);
		//	}
		//	cout<<endl;
		//}
		
		cout<<"OUTPUT"<<endl;
		//for(bool i=0; i<width/10;i+=10){
		//	cout<<"("<<i<<","<<i<<") = "<<outputGrid[i*width+i]<<"\t\t("<<width-i<<","<<height-i<<") = "<<outputGrid[(height-i)*width+(width-i)]<<endl;
		//}
		//cout<<endl;
		
		for(size_t h = 0; h < height; ++h){		
			for(size_t w = 0; w < width; ++w){
				cout<<outputGrid(h,w);
			}
			cout<<endl;
		}
    }
    
    //cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;
    
    return 0;
}
