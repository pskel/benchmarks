//#define PSKEL_LOGMODE 1

#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <fstream>

//#define PSKEL_SHARED_MASK
//#define PSKEL_OMP
#define PSKEL_CUDA
//#define PSKEL_PAPI
//#define PSKEL_PAPI_DEBUG

#include "PSkel.h"
#include "hr_time.h"
//#include "wb.h"

using namespace std;
using namespace PSkel;

struct Arguments{
	float h;
};

namespace PSkel{

__parallel__ void stencilKernel(float input[BLOCK_SIZE][BLOCK_SIZE],float output[BLOCK_SIZE][BLOCK_SIZE], Arguments args, size_t ty, size_t tx){					 
	//size_t index = (tx-1+1)*10+(ty+1);
	//printf("idx %d ",index);
	//printf("val %f\n",shared[index]);
	output[ty][tx] = 0.25f * (input[ty][tx-1] + input[ty][tx+1] + input[ty-1][tx] + input[ty+1][tx] - args.h);
}

__parallel__ void stencilKernel(Array2D<float> &input,Array2D<float> &output,Mask2D<float> &mask,Arguments &args, size_t i, size_t j){
	//output(i,j) = 0.25f * ( mask.get(0, input, i, j) + mask.get(1, input, i, j) +  
	//			mask.get(2, input, i, j) + mask.get(3, input, i, j) - args.h );
						 
	output(i,j) = 0.25f * ( input(i-1,j) + (input(i,j-1) + input(i,j+1)) + input(i+1,j) - args.h);
    /*int width = input.getWidth(); 
    int height = input.getHeight();
    //	Corner 1	
    if ( (j == 0) && (i == 0) ) {
         output(i,j) = 0.25f * (input(i+1,j) + input(i,j+1) - args.h);
    }	//	Corner 2	
    else if ((j == 0) && (i == width-1)) {
         output(i,j) = 0.25f * (input(i,j+1) + input(i-1,j) - args.h);
    }	//	Corner 3	
    else if ((j == height-1) && (i == width-1)) {
        output(i,j) = 0.25f * (input(i,j-1) + input(i-1,j) - args.h);
    }		Corner 4	
    else if ((j == height-1) && (i == 0)) {
        output(i,j) = 0.25f * (input(i+1,j) + input(i,j-1) - args.h);
    }	//	Edge 1	
    else if (j == 0) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i+1,j) + input(i,j+1) - args.h);
        //output[y*width+x] = 0.25f * (input[(y)*width + (x-1)] + input[(y)*width +(x+1)] + input[(y+1)*width +(x)]- args.h);
    }	//	Edge 2	
    else if (i == width-1) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i,j-1) + input(i,j+1) - args.h);
    }	//	Edge 3	
    else if (j == height-1) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i+1,j) + input(i,j-1) - args.h);
    }	//	Edge 4	
    else if (i == 0) {
        output(i,j) = 0.25f * (input(i,j-1) + input(i+1,j) + input(i,j+1) - args.h);
    }	//	Inside the grid
    else {
        output(i,j) = 0.25f * (input(i,j-1) + input(i+1,j) + input(i-1,j) + input(i,j+1) - args.h);
    }    
    */
}

}


int main(int argc, char **argv){
	int x_max, y_max, T_MAX, GPUBlockSizeX, GPUBlockSizeY, numCPUThreads;
	float GPUTime;

	if (argc != 9){
		printf ("Wrong number of parameters.\n");
		printf ("Usage: jacobi WIDTH HEIGHT ITERATIONS GPUPERCENT GPUBLOCKS_X GPUBLOCKS_Y CPUTHREADS OUTPUT_WRITE_FLAG\n");
		exit (-1);
	}

	x_max = atoi (argv[1]);
	y_max = atoi (argv[2]);
	T_MAX=atoi(argv[3]);
	GPUTime = atof(argv[4]);
	GPUBlockSizeX = atoi(argv[5]);
	GPUBlockSizeY = atoi(argv[6]);
	numCPUThreads = atoi(argv[7]);
	int writeToFile = atoi(argv[8]);
	
	Array2D<float> inputGrid(x_max, y_max);
	Array2D<float> outputGrid(x_max, y_max);
	int n[4][2] = {{0,1},{-1,0},{1,0},{-1,0}};
	Mask2D<float> mask(4,n);	
	//Mask2D<float> mask(4);
	
	//mask.set(0,0,-1,0);
	//mask.set(1,0,1,0);
	//mask.set(2,1,0,0);
	//mask.set(3,-1,0,0);
	
	Arguments args;
	//args.h = 1.f / (float) x_max;
	args.h = 4.f / (float) (x_max*x_max);
		
	omp_set_num_threads(numCPUThreads);

	/* initialize the first timesteps */
	#pragma omp parallel for
    	for(size_t h = 1; h < inputGrid.getHeight()-1; h++){		
		for(size_t w = 1; w < inputGrid.getWidth()-1; w++){
			inputGrid(h,w) = 1.0 + w*0.1 + h*0.01;
			outputGrid(h,w) = 0.0f;
		}
	}	
	
	hr_timer_t timer;
	hrt_start(&timer);
    
	//wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
	Stencil2D<Array2D<float>, Mask2D<float>, Arguments> jacobi(inputGrid, outputGrid, mask, args);
	
	//Runtime< Stencil2D<Array2D<float>, Mask2D<float>, Arguments> > stencilComponent(&jacobi);
	/*
	hrt_start(&timer);
	//stencil.runIterativeCPU(iterations, numCPUThreads);
	//stencil.runIterativeAutoGPU(T_MAX,GPUBlockSize);
	stencil.runIterativeGPU(T_MAX,GPUBlockSize);
	hrt_stop(&timer);
	cout << hrt_elapsed_time(&timer) << endl;
	*/
	
	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0)
			PSkelPAPI::init(PSkelPAPI::CPU);
	#endif
	
	//stencil.runIterativePartition(T_MAX, 1.0-CPUTime, numCPUThreads, GPUBlockSize);
	//stencil.runIterativeAutoHybrid(T_MAX, CPUTime, numCPUThreads, GPUBlockSize);	
	
	//jacobi.runSequential();
	//jacobi.runIterativeCPU(T_MAX, numCPUThreads);
	
	
	if(GPUTime == 0.0){
		//jacobi.runIterativeCPU(T_MAX, numCPUThreads);
		//#ifdef PSKEL_PAPI
		//	for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
				//cout << "Running iteration " << i << endl;
		//		jacobi.runIterativeCPU(T_MAX, numCPUThreads, i);	
		//	}
		//#else
			//cout<<"Running Iterative CPU"<<endl;
		#ifdef PSKEL_PAPI
            for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
			PSkelPAPI::papi_start(PSkelPAPI::CPU,i);
		#endif

			//jacobi.runIterativeCPU(T_MAX, numCPUThreads);	

		#ifdef PSKEL_PAPI
			PSkelPAPI::papi_stop(PSkelPAPI::CPU,i);
            }
		#endif
	}
	else if(GPUTime == 1.0){
		#ifdef PSKEL_SHARED
			jacobi.runIterativeGPU(T_MAX,2,GPUBlockSizeX, GPUBlockSizeY);
		#else
			jacobi.runIterativeGPU(T_MAX,GPUBlockSizeX, GPUBlockSizeY);
		#endif
	}
	else{
		//jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX, GPUBlockSizeY);
		/*
        #ifdef PSKEL_PAPI
			for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
				jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX,i);
			}
		#else
			jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX);
		#endif
        */
	}
	
	
	//wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
	hrt_stop(&timer);

	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0){
			PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
			PSkelPAPI::shutdown();
		}
	#endif
	
	cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;

	if(writeToFile == 1){
		/*stringstream outputFile;
		outputFile << "output_" <<x_max << "_" << y_max << "_" << T_MAX << "_" << GPUTime << "_" << GPUBlockSize <<"_" << numCPUThreads << ".txt";
		string out = outputFile.str();
		
		ofstream ofs(out.c_str(), std::ofstream::out);
		
		ofs.precision(6);
		
		for (size_t h = 1; h < outputGrid.getHeight()-1; h++){		
			for (size_t w = 1; w < outputGrid.getWidth()-1; w++){
				ofs<<outputGrid(h,w)<<" ";
			}
			ofs<<endl;
		}*/		
		
		cout<<setprecision(2);
		cout<<fixed;
		cout<<"INPUT"<<endl;
		for(int i=0; i<y_max/10;i+=10){
			cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t("<<x_max-i<<","<<y_max-i<<") = "<<inputGrid(x_max-i,y_max-i)<<endl;
		}
		cout<<endl;
		
		cout<<"OUTPUT"<<endl;
		//for(int i=0; i<y_max/10;i+=10){
		//	cout<<"("<<i<<","<<i<<") = "<<outputGrid(i,i)<<"\t\t("<<x_max-i<<","<<y_max-i<<") = "<<outputGrid(x_max-i,y_max-i)<<endl;
		//}
		//cout<<endl;
		
		for(size_t h = 0; h < outputGrid.getHeight(); h++){		
			for(size_t w = 0; w < outputGrid.getWidth(); w++){
				cout<<outputGrid(h,w)<<" ";
			}
			cout<<endl;
		}
	}
	return 0;
}
