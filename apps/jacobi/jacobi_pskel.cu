#include "hip/hip_runtime.h"
//#define PSKEL_LOGMODE 1

#include <stdio.h>
#include <omp.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <fstream>

//#define PSKEL_SHARED_MASK

#define PSKEL_CUDA

#ifndef PSKEL_OMP
	#ifndef PSKEL_TBB
		#define PSKEL_OMP
		#undef PSKEL_TBB
	#endif
#else 
#ifndef PSKEL_TBB
	#ifndef PSKEL_OMP
		#define PSKEL_TBB
		#undef PSKEL_OMP
	#endif
#endif
#endif

#include "PSkel.h"
#include "hr_time.h"
//#include "wb.h"

using namespace std;
using namespace PSkel;

struct Arguments{
	float h;
};

namespace PSkel{

/*
__parallel__ void stencilKernel(float input[BLOCK_SIZE][BLOCK_SIZE],float output[BLOCK_SIZE][BLOCK_SIZE], Arguments args, size_t ty, size_t tx){					 
	//size_t index = (tx-1+1)*10+(ty+1);
	//printf("idx %d ",index);
	//printf("val %f\n",shared[index]);
	output[ty][tx] = 0.25f * (input[ty][tx-1] + input[ty][tx+1] + input[ty-1][tx] + input[ty+1][tx] - args.h);
}
*/
__parallel__ void stencilKernel(Array2D<float> &input, Array2D<float> &output, const Mask2D<float> &mask, float args, size_t i, size_t j){
	//output(i,j) = 0.25f * ( mask.get(0, input, i, j) + mask.get(1, input, i, j) +  
	//			mask.get(2, input, i, j) + mask.get(3, input, i, j) - args.h );
						 
	
	//float L1 = input(i-1,j);
	//float L2 = input(i,j-1) + input(i,j+1);
	//float L3 = input(i+1,j);

	//output(i,j) = (L1+L2+L3 - args) * 0.25f;
	
	//printf("%f\t",output(i,j));
	output(i,j) = 0.25f * ( input(i-1,j) + (input(i,j-1) + input(i,j+1)) + input(i+1,j) - args);
	
	//output(i,j) = 0.25f * ( input(i-1,j) + (input(i,j-1) + input(i,j+1)) + input(i+1,j) - args);
   /*int width = input.getWidth(); 
    int height = input.getHeight();
    //	Corner 1	
>>>>>>> 94366149f8c7501f5cd79b875bdd69b74eb59bc6
    if ( (j == 0) && (i == 0) ) {
         output(i,j) = 0.25f * (input(i+1,j) + input(i,j+1) - args.h);
    }	//	Corner 2	
    else if ((j == 0) && (i == width-1)) {
         output(i,j) = 0.25f * (input(i,j+1) + input(i-1,j) - args.h);
    }	//	Corner 3	
    else if ((j == height-1) && (i == width-1)) {
        output(i,j) = 0.25f * (input(i,j-1) + input(i-1,j) - args.h);
<<<<<<< HEAD
    }	//	Corner 4	
=======
    }		Corner 4	
>>>>>>> 94366149f8c7501f5cd79b875bdd69b74eb59bc6
    else if ((j == height-1) && (i == 0)) {
        output(i,j) = 0.25f * (input(i+1,j) + input(i,j-1) - args.h);
    }	//	Edge 1	
    else if (j == 0) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i+1,j) + input(i,j+1) - args.h);
        //output[y*width+x] = 0.25f * (input[(y)*width + (x-1)] + input[(y)*width +(x+1)] + input[(y+1)*width +(x)]- args.h);
<<<<<<< HEAD
    }	// Edge 2	
    else if (i == width-1) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i,j-1) + input(i,j+1) - args.h);
        //output[y*width+x] = 0.25f * (input[(y)*width + (x-1)] + input[(y-1)*width +(x)] +input[(y+1)*width +(x)] - args.h);
=======
    }	//	Edge 2	
    else if (i == width-1) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i,j-1) + input(i,j+1) - args.h);
>>>>>>> 94366149f8c7501f5cd79b875bdd69b74eb59bc6
    }	//	Edge 3	
    else if (j == height-1) {
        output(i,j) = 0.25f * (input(i-1,j) + input(i+1,j) + input(i,j-1) - args.h);
    }	//	Edge 4	
    else if (i == 0) {
        output(i,j) = 0.25f * (input(i,j-1) + input(i+1,j) + input(i,j+1) - args.h);
   }	//	Inside the grid  
    else {
        output(i,j) = 0.25f * (input(i,j-1) + input(i+1,j) + input(i-1,j) + input(i,j+1) - args.h);
    }    
    */
  
    }
}


int main(int argc, char **argv){
	int x_max, y_max, T_MAX, pyramidHeight, GPUBlockSizeX, GPUBlockSizeY, numCPUThreads;
	float GPUTime;

	if (argc != 10){
		printf ("Wrong number of parameters.\n");
		printf ("Usage: jacobi WIDTH HEIGHT ITERATIONS PYRAMID_HEIGHT GPUPERCENT GPUBLOCKS_X GPUBLOCKS_Y CPUTHREADS OUTPUT_WRITE_FLAG\n");
		exit (-1);
	}

	x_max = atoi (argv[1]);
	y_max = atoi (argv[2]);
	T_MAX=atoi(argv[3]);
	pyramidHeight=atoi(argv[4]);
	GPUTime = atof(argv[5]);
	GPUBlockSizeX = atoi(argv[6]);
	GPUBlockSizeY = atoi(argv[7]);
	numCPUThreads = atoi(argv[8]);
	int writeToFile = atoi(argv[9]);

	Array2D<float> inputGrid(x_max, y_max);
	Array2D<float> outputGrid(x_max, y_max);
	//int n[4][2] = {{0,1},{-1,0},{1,0},{-1,0}};
	//Mask2D<float> mask(4,n);	
	Mask2D<float> mask(4);
	
	mask.set(0,0,-1,0);
	mask.set(1,0,1,0);
	mask.set(2,1,0,0);
	mask.set(3,-1,0,0);
	
	//Arguments args;
	float args;
	//args.h = 1.f / (float) x_max;
	args = 4.f / (float) (x_max*x_max);
		
	//omp_set_num_threads(numCPUThreads);

	size_t gpuHeight = ceil(inputGrid.getHeight()*GPUTime);
	size_t cpuHeight = inputGrid.getHeight()-gpuHeight;	
	/* initialize the first timesteps */
/*
	#ifdef PSKEL_OMP
	if(GPUTime == 0.0){
		#pragma omp parallel num_threads(numCPUThreads)
    		{
		#pragma omp for
		for(size_t h = 1; h < inputGrid.getHeight()-1; h++){	
			for(size_t w = 1; w < inputGrid.getWidth()-1; w++){
				inputGrid(h,w) = 1.0f + w*0.1 + h*0.01;
				outputGrid(h,w) = 0.0f;
			}
		}
		}
	}
	else{ //NUMA first touch
		//StencilTiling<Array2D<float>,Mask2D<float> > gpuTiling(inputGrid, outputGrid, mask);
		#pragma omp parallel num_threads(numCPUThreads)
    		{
		if(omp_get_thread_num() == 0){
			for(size_t h=0; h < gpuHeight; h++){
				for(size_t w = 0; w < inputGrid.getWidth();w++){
					inputGrid(h,w) = 1.0f; // + w*0.1 + h*0.01;
				 	outputGrid(h,w) = 0.0f;
				}
			}
		}
		else{
			size_t split = ceil(float(cpuHeight)/numCPUThreads);
        		size_t  begin = split * omp_get_thread_num() + gpuHeight;
        		size_t  end = split * (omp_get_thread_num() + 1) + gpuHeight;
                	if(end > inputGrid.getHeight()){
                		end = inputGrid.getHeight();
                	}
	
			for(size_t h = begin; h < end; h++){           
                      		for(size_t w = 0; w < inputGrid.getWidth(); w++){
                                	inputGrid(h,w) = 1.0f; // + w*0.1 + h*0.01;
                                	outputGrid(h,w) = 0.0f;
                        	}
                	}
		}
		}	
	}
	#else
*/	
	//#pragma omp parallel num_threads(numCPUThreads)
	{
	//#pragma omp for 
	for(size_t h = 0; h < y_max; h++){	
		for(size_t w = 0; w < x_max; w++){
			inputGrid(h,w) = 1.0f + w*0.1f + h*0.01f;
			outputGrid(h,w) = 1.0f;
		}
	}
	}
//	#endif
	hr_timer_t timer;
	//hrt_start(&timer);
    
	//wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
	Stencil2D<Array2D<float>, Mask2D<float>, float> jacobi(inputGrid, outputGrid, mask, args);
	
	hrt_start(&timer);
    


	//Runtime< Stencil2D<Array2D<float>, Mask2D<float>, Arguments> > stencilComponent(&jacobi);
	/*
	hrt_start(&timer);
	//stencil.runIterativeCPU(iterations, numCPUThreads);
	//stencil.runIterativeAutoGPU(T_MAX,GPUBlockSize);
	stencil.runIterativeGPU(T_MAX,GPUBlockSize);
	hrt_stop(&timer);
	cout << hrt_elapsed_time(&timer) << endl;
	*/
	
	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0)
			PSkelPAPI::init(PSkelPAPI::CPU);
	#endif
	
	//stencil.runIterativePartition(T_MAX, 1.0-CPUTime, numCPUThreads, GPUBlockSize);
	//stencil.runIterativeAutoHybrid(T_MAX, CPUTime, numCPUThreads, GPUBlockSize);	
	
	//jacobi.runSequential();
	//jacobi.runIterativeCPU(T_MAX, numCPUThreads);
	
	
	if(GPUTime == 0.0){
		//jacobi.runIterativeCPU(T_MAX, numCPUThreads);
		//#ifdef PSKEL_PAPI
		//	for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
				//cout << "Running iteration " << i << endl;
		//		jacobi.runIterativeCPU(T_MAX, numCPUThreads, i);	
		//	}
		//#else
			//cout<<"Running Iterative CPU"<<endl;
		//if(numCPUThreads==1){
		//	cout<<"Running Seq"<<endl;
		//	jacobi.runIterativeSequential(T_MAX);
		//}
		//else{
			#ifdef PSKEL_PAPI
            		for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
				PSkelPAPI::papi_start(PSkelPAPI::CPU,i);
			#endif
			jacobi.runIterativeCPU(T_MAX, numCPUThreads);	
			#ifdef PSKEL_PAPI
				PSkelPAPI::papi_stop(PSkelPAPI::CPU,i);
            		}
			#endif
		//}
	}
	else if(GPUTime == 1.0){
		#ifdef PSKEL_CUDA
		#ifdef PSKEL_SHARED
			jacobi.runIterativeGPU(T_MAX,pyramidHeight,GPUBlockSizeX, GPUBlockSizeY);
		#else
			jacobi.runIterativeGPU(T_MAX,GPUBlockSizeX, GPUBlockSizeY);
		#endif
		#endif
	}
	else{
		#ifdef PSKEL_CUDA
		jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX, GPUBlockSizeY);
		/*
        #ifdef PSKEL_PAPI
			for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
				jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX,i);
			}
		#else
			jacobi.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX);
		#endif
        */
		#endif
	}
	
	
	//wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");
	hrt_stop(&timer);

	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0){
			PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
			PSkelPAPI::shutdown();
		}
	#endif
	
	cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;

	if(writeToFile == 1){
		/*stringstream outputFile;
		outputFile << "output_" <<x_max << "_" << y_max << "_" << T_MAX << "_" << GPUTime << "_" << GPUBlockSize <<"_" << numCPUThreads << ".txt";
		string out = outputFile.str();
		
		ofstream ofs(out.c_str(), std::ofstream::out);
		
		ofs.precision(6);
		
		for (size_t h = 1; h < outputGrid.getHeight()-1; h++){		
			for (size_t w = 1; w < outputGrid.getWidth()-1; w++){
				ofs<<outputGrid(h,w)<<" ";
			}
			ofs<<endl;
		}*/		
		
		cout<<setprecision(2);
		cout<<fixed;
		cout<<"INPUT"<<endl;
		for(size_t h = 0; h < inputGrid.getHeight(); h++){		
			for(size_t w = 0; w < inputGrid.getWidth(); w++){
				cout<<inputGrid(h,w)<<"\t";
			}
			cout<<endl;
		}
		//for(int i=0; i<y_max/10;i+=10){
		//	cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t("<<x_max-i<<","<<y_max-i<<") = "<<inputGrid(x_max-i,y_max-i)<<endl;
		//}
		//cout<<endl;
		
		cout<<"OUTPUT"<<endl;
		//for(int i=0; i<y_max/10;i+=10){
		//	cout<<"("<<i<<","<<i<<") = "<<outputGrid(i,i)<<"\t\t("<<x_max-i<<","<<y_max-i<<") = "<<outputGrid(x_max-i,y_max-i)<<endl;
		//}
		//cout<<endl;
		
		for(size_t h = 0; h < outputGrid.getHeight(); h++){		
			for(size_t w = 0; w < outputGrid.getWidth(); w++){
				cout<<outputGrid(h,w)<<"\t";
			}
			cout<<endl;
		}
	}
	//~inputGrid();
	//~outputGrid();
	//~mask();
	//~jacobi();
	return 0;
}
