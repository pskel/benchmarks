#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>

#ifndef PSKEL_OMP
        #ifndef PSKEL_TBB
                #define PSKEL_OMP
                #undef PSKEL_TBB
        #endif
#else
#ifndef PSKEL_TBB
        #ifndef PSKEL_OMP
                #define PSKEL_TBB
                #undef PSKEL_OMP
        #endif
#endif
#endif

#define PSKEL_CUDA

#include "PSkel.h"
#include "hr_time.h"

#define MASK_RADIUS 2
#define MASK_WIDTH  5

using namespace std;
using namespace PSkel;

//*******************************************************************************************
// CONVOLUTION
//*******************************************************************************************

namespace PSkel{
	__parallel__ void stencilKernel(Array2D<float> &input, Array2D<float> &output, Mask2D<float> &mask, int null, size_t i, size_t j){
		//float accum = 0.0f;
		/*for(int n=0;n<mask.size;n++){
			accum += mask.get(n,input,i,j) * mask.getWeight(n);
		}
		output(i,j)= accum;
		*/

		//float L1 = input(i-2,j-2) * 0.33 + input(i-2,j-1) * 0.33 + input(i-2,j)   * 0.33 + input(i-2,j+1) * 0.33  + input(i-2,j+2) * 0.33;
		float L2 = /*input(i-1,j-2) * 0.33 + */ input(i-1,j-1) * 0.33 + input(i-1,j)   * 0.33 + input(i-1,j+1) * 0.33  /*+ input(i-1,j+2) * 0.33*/; 
        float L3 = /*input(i, j-2)  * 0.33 + */ input(i,j-1)   * 0.33 + input(i,j)     * 0.33 + input(i,j+1)   * 0.33  /*+ input(i, j+2)  * 0.33*/; 
        float L4 = /*input(i+1,j-2) * 0.33 + */ input(i+1,j-1) * 0.33 + input(i+1,j)   * 0.33 + input(i+1,j+1) * 0.33  /*+ input(i+1,j+2) * 0.33*/;  
 		//float L5 = input(i+2,j-2) * 0.33 + input(i+2,j-1) * 0.33 + input(i+2,j)   * 0.33 + input(i+2,j+1) * 0.33  + input(i+2,j+2) * 0.33; 
		
/*
		float L1 = input(i-2,j-2) * 0.33 + input(i-1,j-2) * 0.33 + input(i,j-2)   * 0.33 + input(i+1,j-2) * 0.33  + input(i+2,j-2) * 0.33;
		float L2 = input(i-2,j-1) * 0.33 + input(i-1,j-1) * 0.33 + input(i,j-1)   * 0.33 + input(i+1,j-1) * 0.33  + input(i+2,j-1) * 0.33; 
                float L3 = input(i-2, j)  * 0.33 + input(i-1,j)   * 0.33 + input(i,j)     * 0.33 + input(i+1,j)   * 0.33  + input(i+2, j)  * 0.33; 
                float L4 = input(i-2,j+1) * 0.33 + input(i-1,j+1) * 0.33 + input(i,j+1)   * 0.33 + input(i+1,j+1) * 0.33  + input(i+2,j+1) * 0.33;  
 		float L5 = input(i-2,j+2) * 0.33 + input(i-1,j+2) * 0.33 + input(i,j+2)   * 0.33 + input(i+1,j+2) * 0.33  + input(i+2,j+2) * 0.33; 
		
*/
/*
		float L1 = input(i-2,j-2) * mask.getWeight(0)  + input(i-1,j-2) * mask.getWeight(1)  + input(i,j-2) * mask.getWeight(2)  + input(i+1,j-2) * mask.getWeight(3) + input(i+2,j-2) * mask.getWeight(4);
		float L2 = input(i-2,j-1) * mask.getWeight(5)  + input(i-1,j-1) * mask.getWeight(6)  + input(i,j-1) * mask.getWeight(7)  + input(i+1,j-1) * mask.getWeight(8) + input(i+2,j-1) * mask.getWeight(9); 
                float L3 = input(i-2, j)  * mask.getWeight(10) + input(i-1,j)   * mask.getWeight(11)  + input(i,j)  * mask.getWeight(12)  + input(i+1,j)   * mask.getWeight(13) + input(i+2, j)  * mask.getWeight(14); 
                float L4 = input(i-2,j+1) * mask.getWeight(15) + input(i-1,j+1) * mask.getWeight(16)  + input(i,j+1) * mask.getWeight(17)  + input(i+1,j+1) * mask.getWeight(18) + input(i+2,j+1) * mask.getWeight(19);  
 		float L5 = input(i-2,j+2) * mask.getWeight(20) + input(i-1,j+2) * mask.getWeight(21)  + input(i,j+2) * mask.getWeight(22)  + input(i+1,j+2) * mask.getWeight(23) + input(i+2,j+2) * mask.getWeight(24); 
	*/	
	
		output(i,j) = /*L1 + */L2 + L3 + L4/* + L5*/;
			
		/*output(i,j) = input(i-2,j-2) * 0.33 + input(i-2,j-1) * 0.33 + input(i-2,j)   * 0.33 + input(i-2,j+1) * 0.33  + input(i-2,j+2) * 0.33 + 
			      input(i-1,j-2) * 0.33 + input(i-1,j-1) * 0.33 + input(i-1,j)   * 0.33 + input(i-1,j+1) * 0.33  + input(i-1,j+2) * 0.33 + 
                              input(i, j-2)  * 0.33 + input(i-1,j-1) * 0.33 + input(i,j)     * 0.33 + input(i,j+1)   * 0.33  + input(i,j+2)   * 0.33 + 
                              input(i+1,j-2) * 0.33 + input(i+1,j-1) * 0.33 + input(i+1,j)   * 0.33 + input(i+1,j+1) * 0.33  + input(i+1,j+2) * 0.33 +  
 		 	      input(i+2,j-2) * 0.33 + input(i+2,j-1) * 0.33 + input(i+2,j)   * 0.33 + input(i+2,j+1) * 0.33  + input(i+2,j+2) * 0.33; 
		*/
		/*
		output(i,j) = mask.get(0,input,i,j) * mask.getWeight(0) +
					  mask.get(1,input,i,j) * mask.getWeight(1) +
					  mask.get(2,input,i,j) * mask.getWeight(2) +
					  mask.get(3,input,i,j) * mask.getWeight(3) +
					  mask.get(4,input,i,j) * mask.getWeight(4) +
					  mask.get(5,input,i,j) * mask.getWeight(5) +
					  mask.get(6,input,i,j) * mask.getWeight(6) +
					  mask.get(7,input,i,j) * mask.getWeight(7) +
					  mask.get(8,input,i,j) * mask.getWeight(8) +
					  mask.get(9,input,i,j) * mask.getWeight(9) +
					  mask.get(10,input,i,j) * mask.getWeight(10) +
					  mask.get(11,input,i,j) * mask.getWeight(11) +
					  mask.get(12,input,i,j) * mask.getWeight(12) +
					  mask.get(13,input,i,j) * mask.getWeight(13) +
					  mask.get(14,input,i,j) * mask.getWeight(14) +
					  mask.get(15,input,i,j) * mask.getWeight(15) +
					  mask.get(16,input,i,j) * mask.getWeight(16) +
					  mask.get(17,input,i,j) * mask.getWeight(17) +
					  mask.get(18,input,i,j) * mask.getWeight(18) +
					  mask.get(19,input,i,j) * mask.getWeight(19) +
					  mask.get(20,input,i,j) * mask.getWeight(20) +
					  mask.get(21,input,i,j) * mask.getWeight(21) +
					  mask.get(22,input,i,j) * mask.getWeight(22) +
					  mask.get(23,input,i,j) * mask.getWeight(23) +
					  mask.get(24,input,i,j) * mask.getWeight(24); 
		*/
	}
}//end namespace

//*******************************************************************************************
// MAIN
//*******************************************************************************************

int main(int argc, char **argv){	
		
	Mask2D<float> mask(25);
	float GPUTime;
	int GPUBlockSizeX, GPUBlockSizeY, numCPUThreads,timeTileSize,x_max,y_max;
	
	if (argc != 10){
		printf ("Wrong number of parameters.\n");
		//printf ("Usage: convolution INPUT_IMAGE ITERATIONS GPUTIME GPUBLOCKS CPUTHREADS OUTPUT_WRITE_FLAG\n");
		printf ("Usage: convolution WIDTH HEIGHT ITERATIONS TIME_TILE_SIZE GPUTIME GPUBLOCK_X GPUBLOCK_Y CPUTHREADS OUTPUT_WRITE_FLAG\n");
		printf ("You entered: ");
		for(int i=0; i< argc;i++){
			printf("%s ",argv[i]);
		}
		printf("\n");
		exit (-1);
	}
	
	x_max = atoi(argv[1]);
	y_max = atoi(argv[2]);
	int T_MAX = atoi(argv[3]);
	timeTileSize = atoi(argv[4]);
	GPUTime = atof(argv[5]);
	GPUBlockSizeX = atoi(argv[6]);
	GPUBlockSizeY = atoi(argv[7]);
	numCPUThreads = atoi(argv[8]);
	int writeToFile = atoi(argv[9]);
	
	Array2D<float> inputGrid(x_max, y_max);
	Array2D<float> outputGrid(x_max, y_max);	

	mask.set(0,-2,2,0.1);	mask.set(1,-1,2,0.2);	mask.set(2,0,2,0.3);	mask.set(3,1,2,0.4);	mask.set(4,2,2,0.5);
	mask.set(5,-2,1,0.6);	mask.set(6,-1,1,0.7);	mask.set(7,0,1,0.1);	mask.set(8,1,1,0.3);	mask.set(9,2,1,0.5);
	mask.set(10,-2,0,0.7);	mask.set(11,-1,0,0.1);	mask.set(12,0,0,0.2);	mask.set(13,1,0,0.1);	mask.set(14,2,0,0.5);
	mask.set(15,-2,-1,0.8);	mask.set(16,-1,-1,0.0);	mask.set(17,0,-1,0.1);	mask.set(18,1,-1,0.5);	mask.set(19,2,-1,0.5);
	mask.set(20,-2,-2,0.9);	mask.set(21,-1,-2,0.9);	mask.set(22,0,-2,0.9);	mask.set(23,1,-2,0.9);	mask.set(24,2,-2,0.9);
	
	#pragma omp parallel num_threads(numCPUThreads)
	{
		unsigned int seed = 12345 + 17 *  omp_get_thread_num();
		#pragma omp for
		for (int x = 0; x < x_max; x++){
			for (int y = 0; y < y_max; y++){		
				inputGrid(x,y) = ((float)(rand_r(&seed) % 255))/255;
			}
		}
	}
	cout<<"Data initialized"<<endl;	
	Stencil2D<Array2D<float>, Mask2D<float>, int> stencil(inputGrid, outputGrid, mask, 0);
	hr_timer_t timer;
	
	
	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0)
			PSkelPAPI::init(PSkelPAPI::CPU);
	#endif
	
	hrt_start(&timer);
	
	if(GPUTime == 0.0){
		#ifdef PSKEL_PAPI
		for(unsigned int i=0;i<NUM_GROUPS_CPU;i++){
			PSkelPAPI::papi_start(PSkelPAPI::CPU,i);
		#endif
			stencil.runIterativeCPU(T_MAX, numCPUThreads);
		#ifdef PSKEL_PAPI
			PSkelPAPI::papi_stop(PSkelPAPI::CPU,i);
		}
		#endif
	}
	else if(GPUTime == 1.0){
		stencil.runIterativeGPU(T_MAX, GPUBlockSizeX, GPUBlockSizeY);
	}
	else{
		stencil.runIterativePartition(T_MAX, GPUTime, numCPUThreads,GPUBlockSizeX,GPUBlockSizeY);
	}
	
	hrt_stop(&timer);

	#ifdef PSKEL_PAPI
		if(GPUTime < 1.0){
			PSkelPAPI::print_profile_values(PSkelPAPI::CPU);
			PSkelPAPI::shutdown();
		}
	#endif
	
	cout << "Exec_time\t" << hrt_elapsed_time(&timer) << endl;

	if(writeToFile == 1){
		cout.precision(12);
		cout<<"INPUT"<<endl;
		for(int i=10; i<y_max;i+=10){
			cout<<"("<<i<<","<<i<<") = "<<inputGrid(i,i)<<"\t\t("<<x_max-i<<","<<y_max-i<<") = "<<inputGrid(x_max-i,y_max-i)<<endl;
		}
		cout<<endl;
		
		cout<<"OUTPUT"<<endl;
		for(int i=10; i<y_max;i+=10){
			cout<<"("<<i<<","<<i<<") = "<<outputGrid(i,i)<<"\t\t("<<x_max-i<<","<<y_max-i<<") = "<<outputGrid(x_max-i,y_max-i)<<endl;
		}
		cout<<endl;
	}
	return 0;
}


